#include "hip/hip_runtime.h"

#include<math.h>

/* using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include<hipsparse.h>
#include<hipsparse.h>
#include<hipblas.h>

#include <helper_functions.h>  // helper for shared functions common to cuda samples
#include <hip/hip_runtime_api.h>       // helper function cuda error checking and initialization
#include"header.h"


void solve_matrix_gpu(double **A, double *b, int N, node *no){
	
	//PreConditoned
	double *TempDiagonal;
	TempDiagonal = (double *)malloc(3 * N*sizeof(double));
	for (int i = 0; i < N + N + N; i++){
		TempDiagonal[i] = A[i][i];
	}
	for (int i = 0; i < N + N + N; i++){
		for (int j = 0; j < N + N + N; j++){
			A[i][j] = A[i][j] / sqrt(TempDiagonal[i]);
			A[i][j] = A[i][j] / sqrt(TempDiagonal[j]);
		}
		b[i] = b[i] / sqrt(TempDiagonal[i]);
	}
	
	
	
	//�z�X�g���̃��������m��
	double *hA,*hx,*hb,*hr,*hp,*HostTempVector;
	hA = (double *)malloc(sizeof(double)*(9 * N*N));
	hb = (double *)malloc(sizeof(double)*(N + N + N));
	hx = (double *)malloc(sizeof(double)*(N + N + N));
	hr = (double *)malloc(sizeof(double)*(N + N + N));
	hp = (double *)malloc(sizeof(double)*(N + N + N));
	HostTempVector = (double *)malloc(sizeof(double)*(N + N + N));

	for (int i = 0; i < N + N + N; i++){
		hp[i] = 0;
		hr[i] = 0;
		hp[i] = 0;
		hx[i] = 0;
		hb[i] = 0;
		HostTempVector[i] = 0;
	}


	//�f�[�^���R�s�[
	for (int i = 0; i < N + N+ N; i++){
		for (int j = 0; j < N + N + N; j++){
			hA[i * 3 * N + j] = A[i][j];
		}
	}

	for (int i = 0; i < N; i++){
		hx[i + i + i] = no[i].xd[0];
		hx[i + i + i + 1] = no[i].xd[1];
		hx[i + i + i + 2] = no[i].xd[2];
	}

	for (int i = 0; i < N+N+N; i++){
		hb[i] = b[i];
	}
	

	hipsparseHandle_t cusparseHandle = 0;
	hipsparseMatDescr_t descr = 0;

	hipsparseCreate(&cusparseHandle);


	//�f�o�C�X���̃��������m��
	double *dA, *db, *dx, *dr, *dp, *TempVector;
	double CorrectionCoefficientA = 0;	//�C���W��1
	double CorrectionCoefficientB = 0;	//�C���W��2
	int *npr;
	hipMalloc((void**)&dA, sizeof(double)* 9 * N*N);
	hipMalloc((void**)&db, sizeof(double)* 3 * N);
	hipMalloc((void**)&dx, sizeof(double)* 3 * N);
	hipMalloc((void**)&dr, sizeof(double)* 3 * N);
	hipMalloc((void**)&dp, sizeof(double)* 3 * N);


	hipMalloc((void**)&npr, sizeof(int)* 9 * N*N);
	hipMalloc((void**)&TempVector, sizeof(double)* 3 * N);
	/*Create CUSPARSE context*/
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/*Create CUBLAS context*/
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);


	//�����l��]��
	hipMemcpy(dA, hA, sizeof(double)*(9 * N*N), hipMemcpyHostToDevice);
	hipMemcpy(db, hb, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(dx, hx, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(dp, hp, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(dr, hr, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(TempVector, HostTempVector, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);


	int total;

	hipsparseDnnz(cusparseHandle, HIPSPARSE_DIRECTION_ROW, 3 * N, 3 * N, descr, dA, 3 * N, npr, &total);

	double *csrV_A;
	int *csrC_A;
	int *csrR_A;
	hipMalloc((void**)&csrV_A, sizeof(double)*total);
	hipMalloc((void**)&csrR_A, sizeof(int)* 3 * N + 1);
	hipMalloc((void**)&csrC_A, sizeof(int)*total);

	hipsparseDdense2csr(cusparseHandle, 3 * N, 3 * N, descr, dA, 3 * N, npr, csrV_A, csrR_A, csrC_A);
	double *h_csrV_A;
	h_csrV_A = (double *)malloc(sizeof(double)*total);
	int *h_csrC_A;
	h_csrC_A = (int *)malloc(sizeof(int)* total);
	int *h_csrR_A;
	h_csrR_A = (int *)malloc(sizeof(int)* 3 * N + 1);

	hipMemcpy(h_csrV_A, csrV_A, sizeof(double)*total, hipMemcpyDeviceToHost);
	hipMemcpy(h_csrC_A, csrC_A, sizeof(int)*total, hipMemcpyDeviceToHost);
	hipMemcpy(h_csrR_A, csrR_A, sizeof(int)*(N + N + N), hipMemcpyDeviceToHost);
	printf("total=%d\n", total);
	FILE *fp_V,*fp_C,*fp_R;
	errno_t errorV, errorC, errorR;
	char fnameV[] = "gpuV";
	char fnameC[] = "gpuC";
	char fnameR[] = "gpuR";
	if (errorV = fopen_s(&fp_V, fnameV, "w") != 0){
		printf("\n file open failed \n");
	}

	if (errorC = fopen_s(&fp_C, fnameC, "w") != 0){
		printf("\n file open failed \n");
	}
	
	if (errorR = fopen_s(&fp_R, fnameR, "w") != 0){
		printf("\n file open failed \n");
	}
	for (int i = 0; i < total; i++){
		fprintf(fp_V, "%19.15f\n", h_csrV_A[i]);
		fprintf(fp_C, "%d\n", h_csrC_A[i]);
	}
	for (int i = 0; i < N + N + N + 1; i++){
		fprintf(fp_R, "%d\n", h_csrR_A[i]);
	}
	fclose(fp_V);
	fclose(fp_C);
	fclose(fp_R);

	
	double alpha = -1.0;
	double beta = 1.0;
	hipblasDcopy(cublasHandle, N + N + N, db, 1, TempVector, 1);
	hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, csrV_A, csrR_A, csrC_A, dx, &beta, db);
	hipblasDcopy(cublasHandle, N + N + N, db, 1, dr, 1);
	hipblasDcopy(cublasHandle, N + N + N, TempVector, 1, db, 1);
	hipblasDcopy(cublasHandle, N + N + N, dr, 1, dp, 1);

	double ResidualError = 0;
	hipblasDdot(cublasHandle, N + N + N, dr, 1, dr, 1, &ResidualError);
	printf("iteration:\t%d\tresidual error:\t%1.20f\n", 0, ResidualError);
	

	int Iteration = 1;
	double error[1000000] = {};
	error[0] = ResidualError;
	clock_t start, end;
	start = clock();

	while (1){
		double Denominator = 0.0f;	//����
		double Numerator = 0.0f;		//���q

		hipblasDdot(cublasHandle, N + N + N, dp, 1, dr, 1, &Numerator);
	
		alpha = 1.0;
		beta = 0.0f;
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, csrV_A, csrR_A, csrC_A, dp, &beta, TempVector);
		hipblasDdot(cublasHandle, N + N + N, dp, 1, TempVector, 1, &Denominator);
		
		CorrectionCoefficientA = Numerator / Denominator;
		
		hipblasDaxpy(cublasHandle, N + N + N, &CorrectionCoefficientA, dp, 1, dx, 1);
	
		alpha = -CorrectionCoefficientA;
		beta = 1;
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, csrV_A, csrR_A, csrC_A, dp, &beta, dr);
		
		hipblasDdot(cublasHandle, N + N + N, dr, 1, dr, 1, &ResidualError);
			
		Iteration++;
		error[Iteration] = ResidualError;
		//if (ResidualError < EPS*EPS){
		//	double  stability = fabs(error[Iteration] - error[Iteration - 1000]);
		//	if (stability < EPS*EPS / 10){
		//		break;		//EPS�̔��f
		//	}
		//}
		if (ResidualError == 0)break;


		alpha = 1.0f;
		beta = 0.0f;
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, csrV_A, csrR_A, csrC_A, dp, &beta, TempVector);
		hipblasDdot(cublasHandle, N + N + N, dr, 1, TempVector, 1, &Numerator);
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, csrV_A, csrR_A, csrC_A, dp, &beta, TempVector);
		hipblasDdot(cublasHandle, N + N + N, dp, 1, TempVector, 1, &Denominator);
		CorrectionCoefficientB = -Numerator / Denominator;
		hipblasDscal(cublasHandle, N + N + N, &CorrectionCoefficientB, dp, 1);
		CorrectionCoefficientB = 1;
		hipblasDaxpy(cublasHandle, N + N + N,&CorrectionCoefficientB , dr, 1, dp, 1);
	}
	end = clock();
	printf("��������-CG�@%d\n", end - start);
	hipMemcpy(hx, dx, sizeof(double)*(N + N + N), hipMemcpyDeviceToHost);
	for (int i = 0; i < N + N + N; i++){
		b[i] = hx[i] / sqrt(TempDiagonal[i]);
	}
}




void solve_matrix_gpu_CSR(double *CSR_Kval,int *CSR_col,int *CSR_row, double *b, int N, node *no,int RealNumberOfValues){

	//Precondtioned
	double *TempDiagonal;
	TempDiagonal = (double *)malloc(3 * N*sizeof(double));
	for (int i = 0; i < N + N + N; i++){
		for (int j = CSR_row[i]; j < CSR_row[i+1]; j++){
			if (CSR_col[j] == i)TempDiagonal[i] = CSR_Kval[j];
		}
	}





	//PreConditoned
	//double *TempDiagonal;
	//TempDiagonal = (double *)malloc(3 * N*sizeof(double));
	//for (int i = 0; i < N + N + N; i++){
	//	TempDiagonal[i] = A[i][i];
	//}
	//for (int i = 0; i < N + N + N; i++){
	//	for (int j = 0; j < N + N + N; j++){
	//		A[i][j] = A[i][j] / sqrt(TempDiagonal[i]);
	//		A[i][j] = A[i][j] / sqrt(TempDiagonal[j]);
	//	}
	//	b[i] = b[i] / sqrt(TempDiagonal[i]);
	//}
	//
	
	
	//�z�X�g���̃��������m��
	double *h_CSR_Kval,*hx,*hb,*hr,*hp,*HostTempVector;
	int *h_CSR_col, *h_CSR_row;
	h_CSR_Kval = (double *)malloc(sizeof(double)*(RealNumberOfValues));
	h_CSR_col = (int *)malloc(sizeof(int)*(RealNumberOfValues));
	h_CSR_row = (int *)malloc(sizeof(int)*(N + N + N));
	hb = (double *)malloc(sizeof(double)*(N + N + N));
	for (int i = 0; i < RealNumberOfValues; i++){
		h_CSR_Kval[i] = CSR_Kval[i];
	}

	//�Ίp�X�P�[�����O
	for (int i = 0; i < N + N + N; i++){
		for (int j = CSR_row[i]; j < CSR_row[i + 1]; j++){
			h_CSR_Kval[j] = h_CSR_Kval[j] / sqrt(TempDiagonal[i]);
			h_CSR_Kval[j] = h_CSR_Kval[j] / sqrt(TempDiagonal[CSR_col[j]]);
		}
	}

	for (int i = 0; i < RealNumberOfValues; i++){
		h_CSR_col[i] = CSR_col[i];
	}
	for (int i = 0; i < N + N + N ; i++){
		h_CSR_row[i] = CSR_row[i];
	}
	for (int i = 0; i < N + N + N; i++){
		hb[i] = b[i] / sqrt(TempDiagonal[i]);
	}

	FILE *fp_V, *fp_C, *fp_R;
	errno_t errorV, errorC, errorR;
	char fnameV[] = "cpuV";
	char fnameC[] = "cpuC";
	char fnameR[] = "cpuR";
	if (errorV = fopen_s(&fp_V, fnameV, "w") != 0){
		printf("\n file open failed \n");
	}

	if (errorC = fopen_s(&fp_C, fnameC, "w") != 0){
		printf("\n file open failed \n");
	}

	if (errorR = fopen_s(&fp_R, fnameR, "w") != 0){
		printf("\n file open failed \n");
	}
	for (int i = 0; i < RealNumberOfValues; i++){
		fprintf(fp_V, "%19.15f\n", h_CSR_Kval[i]);
		fprintf(fp_C, "%d\n", h_CSR_col[i]);
	}
	for (int i = 0; i < N + N + N; i++){
		fprintf(fp_R, "%d\n", h_CSR_row[i]);
	}
	fclose(fp_V);
	fclose(fp_C);
	fclose(fp_R);


	hx = (double *)malloc(sizeof(double)*(N + N + N));
	hr = (double *)malloc(sizeof(double)*(N + N + N));
	hp = (double *)malloc(sizeof(double)*(N + N + N));
	HostTempVector = (double *)malloc(sizeof(double)*(N + N + N));

	for (int i = 0; i < N + N + N; i++){
		hp[i] = 0;
		hr[i] = 0;
		hx[i] = 0;
		HostTempVector[i] = 0;
	}


	//�f�[�^���R�s�[
	//for (int i = 0; i < N; i++){
	//	hx[i + i + i] = no[i].xd[0];
	//	hx[i + i + i + 1] = no[i].xd[1];
	//	hx[i + i + i + 2] = no[i].xd[2];
	//}


	

	hipsparseHandle_t cusparseHandle = 0;
	hipsparseMatDescr_t descr = 0;

	hipsparseCreate(&cusparseHandle);


	//�f�o�C�X���̃��������m��
	double *d_CSR_Kval,*db, *dx, *dr, *dp, *TempVector;
	int *d_CSR_col, *d_CSR_row;
	double CorrectionCoefficientA = 0;	//�C���W��1
	double CorrectionCoefficientB = 0;	//�C���W��2
	int *npr;

	hipMalloc((void**)&d_CSR_Kval, sizeof(double)*RealNumberOfValues);
	hipMalloc((void**)&d_CSR_col, sizeof(int)*RealNumberOfValues);
	hipMalloc((void**)&d_CSR_row, sizeof(int)*(N + N + N )+1);
	hipMalloc((void**)&db, sizeof(double)* 3 * N);
	hipMalloc((void**)&dx, sizeof(double)* 3 * N);
	hipMalloc((void**)&dr, sizeof(double)* 3 * N);
	hipMalloc((void**)&dp, sizeof(double)* 3 * N);


	hipMalloc((void**)&npr, sizeof(int)* 9 * N*N);
	hipMalloc((void**)&TempVector, sizeof(double)* 3 * N);
	/*Create CUSPARSE context*/
	hipsparseCreateMatDescr(&descr);
	hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

	/*Create CUBLAS context*/
	hipblasHandle_t cublasHandle = 0;
	hipblasStatus_t hipblasStatus_t;
	hipblasStatus_t = hipblasCreate(&cublasHandle);


	//�����l��]��
	hipMemcpy(d_CSR_Kval, h_CSR_Kval, sizeof(double)*RealNumberOfValues, hipMemcpyHostToDevice);
	hipMemcpy(d_CSR_col, h_CSR_col, sizeof(int)*RealNumberOfValues, hipMemcpyHostToDevice);
	hipMemcpy(d_CSR_row, h_CSR_row, sizeof(int)*(N + N + N ), hipMemcpyHostToDevice);
	hipMemcpy(db, hb, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(dx, hx, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(dp, hp, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(dr, hr, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);
	hipMemcpy(TempVector, HostTempVector, sizeof(double)*(N + N + N), hipMemcpyHostToDevice);


	int total=RealNumberOfValues;


	//double *csrV_A;
	//int *csrC_A;
	//int *csrR_A;
	//hipMalloc((void**)&csrV_A, sizeof(double)*total);
	//hipMalloc((void**)&csrR_A, sizeof(int)* 3 * N + 1);
	//hipMalloc((void**)&csrC_A, sizeof(int)*total);

	
	double alpha = -1.0;
	double beta = 1.0;
	hipblasDcopy(cublasHandle, N + N + N, db, 1, TempVector, 1);
	hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, d_CSR_Kval, d_CSR_row, d_CSR_col, dx, &beta, db);
	hipblasDcopy(cublasHandle, N + N + N, db, 1, dr, 1);
	hipblasDcopy(cublasHandle, N + N + N, TempVector, 1, db, 1);
	hipblasDcopy(cublasHandle, N + N + N, dr, 1, dp, 1);

	double ResidualError = 0;
	hipblasDdot(cublasHandle, N + N + N, dr, 1, dr, 1, &ResidualError);
	//printf("iteration:\t%d\tresidual error:\t%1.20f\n", 0, ResidualError);
	

	int Iteration = 1;
	double error[1000000] = {};
	error[0] = ResidualError;
	clock_t start, end;
	start = clock();

	while (1){
		double Denominator = 0.0f;	//����
		double Numerator = 0.0f;		//���q

		hipblasDdot(cublasHandle, N + N + N, dp, 1, dr, 1, &Numerator);
	
		alpha = 1.0;
		beta = 0.0f;
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, d_CSR_Kval, d_CSR_row, d_CSR_col, dp, &beta, TempVector);
		hipblasDdot(cublasHandle, N + N + N, dp, 1, TempVector, 1, &Denominator);
		
		CorrectionCoefficientA = Numerator / Denominator;
		
		hipblasDaxpy(cublasHandle, N + N + N, &CorrectionCoefficientA, dp, 1, dx, 1);
	
		alpha = -CorrectionCoefficientA;
		beta = 1;	
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, d_CSR_Kval, d_CSR_row, d_CSR_col, dp, &beta, dr);
		hipblasDdot(cublasHandle, N + N + N, dr, 1, dr, 1, &ResidualError);
			
		Iteration++;
		error[Iteration] = ResidualError;
		//printf("iteration:\t%d\tresidual error:\t%1.20f\n", Iteration, error[Iteration]);
		//if (ResidualError < EPS*EPS){
		//	double  stability = fabs(error[Iteration] - error[Iteration - 1000]);
		//	if (stability < EPS*EPS / 10){
		//		break;		//EPS�̔��f
		//	}
		//}
		if (ResidualError <= 1e-15f*1e-15f)break;


		alpha = 1.0f;
		beta = 0.0f;
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, d_CSR_Kval, d_CSR_row, d_CSR_col, dp, &beta, TempVector);
		hipblasDdot(cublasHandle, N + N + N, dr, 1, TempVector, 1, &Numerator);
		hipsparseDcsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3 * N, 3 * N, total, &alpha, descr, d_CSR_Kval, d_CSR_row, d_CSR_col, dp, &beta, TempVector);
		hipblasDdot(cublasHandle, N + N + N, dp, 1, TempVector, 1, &Denominator);
		CorrectionCoefficientB = -Numerator / Denominator;
		hipblasDscal(cublasHandle, N + N + N, &CorrectionCoefficientB, dp, 1);
		CorrectionCoefficientB = 1;
		hipblasDaxpy(cublasHandle, N + N + N,&CorrectionCoefficientB , dr, 1, dp, 1);
	}
	end = clock();
	printf("��������-CG�@%d\n", end - start);
	hipMemcpy(hx, dx, sizeof(double)*(N + N + N), hipMemcpyDeviceToHost);
	for (int i = 0; i < N + N + N; i++){
		b[i] = hx[i] / sqrt(TempDiagonal[i]);
	}

	FILE *fp_errors;
	errno_t errors;
	char file_name[100] = {};
	printf("FILE NAME:");
	scanf("%s", file_name);
	if (errors = fopen_s(&fp_errors, file_name, "w") != 0){
		printf("\n file open failed \n");
	}
	for (int i = 0; i < Iteration; i++){
		fprintf(fp_errors, "%d,%31.30f\n", i,error[i]);
	}
	fclose(fp_errors);
}

